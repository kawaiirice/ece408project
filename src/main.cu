#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstddef>
#include <iostream>
#include <numeric>
#include <map>
#include <sys/time.h>
#include <valarray>

#include <hdf5.h>

#include "range.hpp"
#include "utils.hpp"

#define NUM_ROWS 28
#define NUM_COLS 28
#define NUM_CHANNELS 1
#define NUM_DIGITS 10

static int FLAGS_batch_size = 10000;
static std::string FLAGS_testdata{};
static std::string FLAGS_model{};

// Data and reference data dimensions
static int xdims[] = {FLAGS_batch_size, NUM_ROWS, NUM_COLS, NUM_CHANNELS};
static int rdims[] = {FLAGS_batch_size, NUM_DIGITS};

// Model dimensions
static int conv1dims[] = {5, 5, 1, 32};
static int conv2dims[] = {5, 5, 32, 64};
static int fc1dims[]   = {1024, 128};
static int fc2dims[]   = {128, 10};

static int loadData(float *x, float *y) {
  // Open the data file
  const auto file_id =
      H5Fopen(FLAGS_testdata.c_str(), H5F_ACC_RDWR, H5P_DEFAULT);

  // Open the dataset x and y
  const auto x_id = H5Dopen2(file_id, "/x", H5P_DEFAULT);
  const auto y_id = H5Dopen2(file_id, "/y", H5P_DEFAULT);

  // Get the dataset x dimensions
  const auto xspace = H5Dget_space(x_id);
  const auto xndims = H5Sget_simple_extent_ndims(xspace);
  assert(xndims == 4);

  hsize_t input_dims[xndims];
  H5Sget_simple_extent_dims(xspace, input_dims, NULL);
  if (input_dims[0] != FLAGS_batch_size) {
    std::cout << "data size does not match batch size specified!\n";
    return 1; // return error
  }
  std::cout << "input dimensions = " << input_dims[0] << " x " << input_dims[1]
            << " x " << input_dims[2] << " x " << input_dims[3] << "\n";

  // Read the dataset x and y
  check_success(
      H5Dread(x_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, x));
  check_success(
      H5Dread(y_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, y));

  // Close the dataset x and y
  check_success(H5Dclose(x_id));
  check_success(H5Dclose(y_id));

  // Close the file
  check_success(H5Fclose(file_id));

  // return success
  return 0;
}

static void loadModel(float *conv1, float *conv2, float *fc1, float *fc2) {
  // Open the model file
  const auto file_id = H5Fopen(FLAGS_model.c_str(), H5F_ACC_RDWR, H5P_DEFAULT);

  // Open the dataset
  const auto conv1_id = H5Dopen2(file_id, "/conv1", H5P_DEFAULT);
  const auto conv2_id = H5Dopen2(file_id, "/conv2", H5P_DEFAULT);
  const auto fc1_id   = H5Dopen2(file_id, "/fc1", H5P_DEFAULT);
  const auto fc2_id   = H5Dopen2(file_id, "/fc2", H5P_DEFAULT);

  // Read the dataset
  check_success(H5Dread(conv1_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL,
                        H5P_DEFAULT, conv1));
  check_success(H5Dread(conv2_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL,
                        H5P_DEFAULT, conv2));
  check_success(
      H5Dread(fc1_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, fc1));
  check_success(
      H5Dread(fc2_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, fc2));

  // Close the dataset x and y
  check_success(H5Dclose(conv1_id));
  check_success(H5Dclose(conv2_id));
  check_success(H5Dclose(fc1_id));
  check_success(H5Dclose(fc2_id));

  // Close the file
  check_success(H5Fclose(file_id));
}

// // From book chapter Figure 16.4
// static void conv_forward_valid(const float *X, const int xdims[4],
//                                const float *W, const int wdims[4], float *Y,
//                                const int ydims[4]) {
//   const auto filter_h   = wdims[0];
//   const auto filter_w   = wdims[1];
//   const auto in_channel = wdims[2];
//
//   for (const auto i : range(0, ydims[0])) {
//     for (const auto m : range(0, ydims[3])) {
//       for (const auto w : range(0, ydims[2])) {
//         for (const auto h : range(0, ydims[1])) {
//           for (const auto p : range(0, filter_h)) {
//             for (const auto q : range(0, filter_w)) {
//               for (const auto c : range(0, in_channel)) {
//                 const auto yoffset =
//                     ((i * ydims[1] + h) * ydims[2] + w) * ydims[3] + m;
//                 const auto xoffset = i * xdims[1] * xdims[2] * xdims[3] +
//                                      (h + p) * xdims[2] * xdims[3] +
//                                      (w + q) * xdims[3] + c;
//                 const auto woffset = p * wdims[1] * wdims[2] * wdims[3] +
//                                      q * wdims[2] * wdims[3] + c * wdims[3] + m;
//                 Y[yoffset] += X[xoffset] * W[woffset];
//               }
//             }
//           }
//         }
//       }
//     }
//   }
// }

__global__ void conv_forward_valid_kernel(const float *X, const int xdims[4],
                               const float *W, const int wdims[4], float *Y,
                               const int ydims[4]) {
	const auto filter_h   = wdims[0];
	const auto filter_w   = wdims[1];
	const auto in_channel = wdims[2];

	int i = blockIdx.x;
	int m = blockIdx.y;
	int idx = blockIdx.z*blockDim.x + threadIdx.x;
	int h = idx/ydims[2];
	int w = idx%ydims[2];
	if(h < ydims[1] && w < ydims[2]){
    // float acc = 0;
		const auto yoffset = ((i * ydims[1] + h) * ydims[2] + w) * ydims[3] + m;
		for (const auto p : range(0, filter_h)) {
			for (const auto q : range(0, filter_w)) {
				for (const auto c : range(0, in_channel)) {
				const auto xoffset = i * xdims[1] * xdims[2] * xdims[3] +
								 (h + p) * xdims[2] * xdims[3] +
								 (w + q) * xdims[3] + c;
				const auto woffset = p * wdims[1] * wdims[2] * wdims[3] +
								 q * wdims[2] * wdims[3] + c * wdims[3] + m;
				Y[yoffset] += X[xoffset] * W[woffset];
        // acc += X[xoffset] * W[woffset];
				}
			}
		}
    // Y[yoffset] = acc;
	}
}

// // Recified linear unit 4d
// static void relu4(float *X, const int xdims[4]) {
//   for (const auto i : range(0, xdims[0] * xdims[1] * xdims[2] * xdims[3])) {
//     X[i] = (X[i] < 0) ? 0 : X[i];
//   }
// }

__global__ void relu4_kernel(float *X, const int xdims[4]){
	int row = blockIdx.x*blockDim.x+threadIdx.x;
	if(row < xdims[0] * xdims[1] * xdims[2] * xdims[3])
		X[row] = (X[row] < 0) ? 0 : X[row];
}

// // Recified linear unit 2d
// static void relu2(float *X, const int xdims[2]) {
//   for (const auto i : range(0, xdims[0] * xdims[1])) {
//     X[i] = (X[i] < 0) ? 0 : X[i];
//   }
// }

__global__ void relu2_kernel(float *X, const int xdims[2]){
 	int i = blockIdx.x*blockDim.x+threadIdx.x;
 	if(i < xdims[0] * xdims[1])
 		X[i] = (X[i] < 0) ? 0 : X[i];
}

// // From book chapter Figure 16.5
// static void average_pool(const float *X, const int xdims[4],
//                          const int pool_size, float *Y, const int ydims[4]) {
//   for (const auto i : range(0, ydims[0])) {
//     for (const auto m : range(0, ydims[3])) {
//       for (const auto w : range(0, ydims[2])) {
//         for (const auto h : range(0, ydims[1])) {
//           for (const auto p : range(0, pool_size)) {
//             for (const auto q : range(0, pool_size)) {
//               const auto yoffset =
//                   ((i * ydims[1] + h) * ydims[2] + w) * ydims[3] + m;
//               const auto xoffset = i * xdims[1] * xdims[2] * xdims[3] +
//                                    (pool_size * h + p) * xdims[2] * xdims[3] +
//                                    (pool_size * w + q) * xdims[3] + m;
//               Y[yoffset] += X[xoffset] / (1.0f * pool_size * pool_size);
//             }
//           }
//         }
//       }
//     }
//   }
// }

__global__ void average_pool_kernel(const float *X, const int xdims[4],
                         const int pool_size, float *Y, const int ydims[4], const int w_grid) {
	int i = blockIdx.x;
	int m = blockIdx.y;
	int idx = blockIdx.z*blockDim.x + threadIdx.x;
	int h = idx/ydims[2];
	int w = idx%ydims[2];
	//int h = blockIdx.z/(w_grid) + threadIdx.y;
	//int w = blockIdx.z%(w_grid) + threadIdx.x;
	if(h < ydims[1] && w < ydims[2]){
	  const auto yoffset = ((i * ydims[1] + h) * ydims[2] + w) * ydims[3] + m;
	  //float acc = 0;
	  for (const auto p : range(0, pool_size)) {
		for (const auto q : range(0, pool_size)) {
		  const auto xoffset = i * xdims[1] * xdims[2] * xdims[3] +
							   (pool_size * h + p) * xdims[2] * xdims[3] +
							   (pool_size * w + q) * xdims[3] + m;
		  //acc += X[xoffset] / (1.0f * pool_size * pool_size);
		  Y[yoffset] += X[xoffset] / (1.0f * pool_size * pool_size);
		  //atomicAdd(&Y[yoffset], X[xoffset]/(1.0f * pool_size * pool_size));
		}
	  }
	  //Y[yoffset] = acc;
	}
}

// static void fully_forward(const float *X, const int xdims[2], float *W,
//                           const int wdims[2], float *Y, const int ydims[2]) {
//   for (const auto i : range(0, xdims[0])) {
//     for (const auto j : range(0, wdims[1])) {
//       float sum = 0;
//       for (const auto k : range(0, xdims[1])) {
//         sum += X[i * xdims[1] + k] * W[k * wdims[1] + j];
//       }
//       Y[i * wdims[1] + j] = sum;
//     }
//   }
// }

__global__ void fully_forward_kernel (const float *X, const int xdims[2], float *W,
                          const int wdims[2], float *Y, const int ydims[2]) {
    int i, j;
    i = blockIdx.y * blockDim.y + threadIdx.y; // y index of output
    j = blockIdx.x * blockDim.x + threadIdx.x; // x index of output

    if(i < ydims[0] && j < ydims[1]){
      float sum = 0;
      for (const auto k : range(0, xdims[1])) {
        sum += X[i * xdims[1] + k] * W[k * wdims[1] + j];
      }
      Y[i * wdims[1] + j] = sum;
    }
}

// Choose the guess with largest score
static void argmax(const float *X, const int xdims[2], int *Y) {
  for (const auto i : range(0, xdims[0])) {
    auto max_idx = 0;
    auto max     = X[i * xdims[1]];
    for (const auto j : range(0, xdims[1])) {
      const auto elem = X[(i * xdims[1]) + j];
      if (elem > max) {
        max_idx = j;
        max     = elem;
      }
    }
    Y[i] = max_idx;
  }
}

__global__ void argmax_kernel(const float *X, const int xdims[2], int *Y) {

  int i= blockIdx.x * blockDim.x + threadIdx.x;
  
  if(i < xdims[0]) {
    auto max_idx = 0;
    auto max = X[i * xdims[1]];
    for (const auto j : range(0, xdims[1])) {
      const auto elem = X[(i * xdims[1]) + j];
      if (elem > max) {
        max_idx = j;
        max     = elem;
      }
    }
    Y[i] = max_idx;
  }

}

// Forward operation for the CNN, a combination of conv layer + average pooling
// + relu
void forward_operation(float *x, float *conv1, float *conv2, float *fc1,
                       float *fc2, int *out) {
  // conv layer
  const int adims[] = {xdims[0], (xdims[1] - conv1dims[0] + 1),
                       (xdims[2] - conv1dims[1] + 1), conv1dims[3]};
  int device_a_size = adims[0]*adims[1]*adims[2]*adims[3];

  const int pool_size = 2;
  const int bdims[]   = {adims[0], adims[1] / pool_size, adims[2] / pool_size, adims[3]};
  int device_b_size = bdims[0]*bdims[1]*bdims[2]*bdims[3];

  const int cdims[] = {bdims[0], (bdims[1] - conv2dims[0] + 1),
                       (bdims[2] - conv2dims[1] + 1), conv2dims[3]};
  int device_c_size = cdims[0]*cdims[1]*cdims[2]*cdims[3];

  auto a = zeros<float>(adims);

  float *device_x;
  int device_x_size = xdims[0] * xdims[1] * xdims[2] * xdims[3];

  hipMalloc((void **)&device_x, sizeof(float)*device_b_size);
  hipMemcpy(device_x, x, sizeof(float)*device_x_size, hipMemcpyHostToDevice);

  int *device_xdims;
  hipMalloc((void **)&device_xdims, sizeof(int)*4);
  hipMemcpy(device_xdims, xdims, sizeof(int)*4, hipMemcpyHostToDevice);

  float *device_conv1;
  int device_conv1_size = conv1dims[0] * conv1dims[1] * conv1dims[2] * conv1dims[3];
  int device_conv2_size = conv2dims[0] * conv2dims[1] * conv2dims[2] * conv2dims[3];
  hipMalloc((void **)&device_conv1, sizeof(float)*fc1dims[0] * fc1dims[1]);
  hipMemcpy(device_conv1, conv1, sizeof(float)*device_conv1_size, hipMemcpyHostToDevice);

  int *device_conv1dims;
  hipMalloc((void **)&device_conv1dims, sizeof(int)*4);
  hipMemcpy(device_conv1dims, conv1dims, sizeof(int)*4, hipMemcpyHostToDevice);

  float *device_a;
  int *device_adims;

  hipMalloc((void **)&device_adims, sizeof(int)*4);
  hipMemcpy(device_adims, adims, sizeof(int)*4, hipMemcpyHostToDevice);

  hipMalloc((void **)&device_a, sizeof(float)* device_a_size);
  hipMemcpy(device_a, a, sizeof(float)*device_a_size, hipMemcpyHostToDevice);

  //conv_forward_valid(x, xdims, conv1, conv1dims, a, adims);
  dim3 DimGrid0(adims[0], adims[3], ceil(adims[1]*adims[2]/256.0));
  dim3 DimBlock0(256, 1, 1);

  const auto start = now();
  conv_forward_valid_kernel<<<DimGrid0, DimBlock0>>>(device_x, device_xdims, device_conv1, device_conv1dims, device_a, device_adims);
  const auto end = now();

  //hipMemcpy(a, device_a, sizeof(float)*device_a_size, hipMemcpyDeviceToHost);
  const auto elapsed = std::chrono::duration<double, std::milli>(end - start).count();
  std::cout << "Done with " << adims[0] << " queries in Conv1 "
          << "elapsed = " << elapsed << " milliseconds." << "\n";

  /// relu layer
  //relu4(a, adims);

  // relu kernel start layer
  /*
  float *device_a;
  int *device_adims;
  int device_a_size = adims[0]*adims[1]*adims[2]*adims[3];

  hipMalloc((void **)&device_adims, sizeof(int)*4);
  hipMemcpy(device_adims, adims, sizeof(int)*4, hipMemcpyHostToDevice);

  hipMalloc((void **)&device_a, sizeof(float)* device_a_size);
  hipMemcpy(device_a, a, sizeof(float)*device_a_size, hipMemcpyHostToDevice);
  */

  dim3 DimGrid(ceil(device_a_size/256), 1, 1);
  dim3 DimBlock(256, 1, 1);
  relu4_kernel<<<DimGrid, DimBlock>>>(device_a, device_adims);

  //hipMemcpy(device_adims, adims, sizeof(int)*4, hipMemcpyDeviceToHost);
  //hipMemcpy(a, device_a, sizeof(float)*device_a_size, hipMemcpyDeviceToHost);
  // relu kernel end layer



  // average pooling
  //const int pool_size = 2;
  //const int bdims[]   = {adims[0], adims[1] / pool_size, adims[2] / pool_size, adims[3]};
  auto b = zeros<float>(bdims);
  //average_pool(a, adims, pool_size, b, bdims);

  //std::cout << "Device a and c sizes: " << device_a_size << " " << device_c_size << std::endl;

  hipMemcpy(device_xdims, bdims, sizeof(int)*4, hipMemcpyHostToDevice);

  hipMemcpy(device_x, b, sizeof(float)*device_b_size, hipMemcpyHostToDevice);

  int tile_size = 16;
  int w_grid = ceil(bdims[2]/tile_size);
  int h_grid = ceil(bdims[1]/tile_size);

  dim3 DimGrid1(bdims[0], bdims[3], ceil(bdims[1]*bdims[2]/256.0));
  dim3 DimBlock1(256, 1, 1);

  const auto start2 = now();
  average_pool_kernel<<<DimGrid1, DimBlock1>>>(device_a, device_adims, pool_size, device_x, device_xdims, w_grid);
  const auto end2 = now();
  const auto elapsed2 = std::chrono::duration<double, std::milli>(end2 - start2).count();
  std::cout << "Done with " << adims[0] << " queries in pool1 "
          << "elapsed = " << elapsed2 << " milliseconds." << "\n";
  //hipMemcpy(b, device_x, sizeof(float)*device_b_size, hipMemcpyDeviceToHost);

  /*
  // average pooling kernel start
  float *device_b;
  int *device_bdims;
  int device_b_size = bdims[0]*bdims[1]*bdims[2]*bdims[3];

  hipMalloc((void **)&device_bdims, sizeof(int)*4);
  hipMemcpy(device_bdims, bdims, sizeof(int)*4, hipMemcpyHostToDevice);

  hipMalloc((void **)&device_b, sizeof(float)* device_b_size);
  hipMemcpy(device_b, b, sizeof(float)*device_b_size, hipMemcpyHostToDevice);

  int tile_size = 16;
  int w_grid = ceil(bdims[2]/tile_size);
  int h_grid = ceil(bdims[1]/tile_size);
  //dim3 DimGrid1(bdims[0], bdims[3], w_grid*h_grid);
  dim3 DimGrid1(bdims[0], bdims[3], ceil(bdims[1]*bdims[2]/256.0));
  //dim3 DimBlock1(tile_size, tile_size, 1);
  dim3 DimBlock1(256, 1, 1);
  average_pool_kernel<<<DimGrid1, DimBlock1>>>(device_a, device_adims, pool_size, device_b, device_bdims, w_grid);

  hipMemcpy(device_bdims, bdims, sizeof(int)*4, hipMemcpyDeviceToHost);
  hipMemcpy(b, device_b, sizeof(float)*device_b_size, hipMemcpyDeviceToHost);
  // average pooling kernel end
  */


  // conv layer

  //const int cdims[] = {bdims[0], (bdims[1] - conv2dims[0] + 1),
   //                    (bdims[2] - conv2dims[1] + 1), conv2dims[3]};
  //int device_c_size = cdims[0]*cdims[1]*cdims[2]*cdims[3];
  auto c = zeros<float>(cdims);
  //conv_forward_valid(b, bdims, conv2, conv2dims, c, cdims);

  // kernel conv start
  // device_x holds device_b, device_conv1 holds device_conv2, device_a holds c

  hipMemcpy(device_conv1, conv2, sizeof(float)*device_conv2_size, hipMemcpyHostToDevice);

  hipMemcpy(device_conv1dims, conv2dims, sizeof(int)*4, hipMemcpyHostToDevice);

  hipMemcpy(device_adims, cdims, sizeof(int)*4, hipMemcpyHostToDevice);

  hipMemcpy(device_a, c, sizeof(float)*device_c_size, hipMemcpyHostToDevice);

  //conv_forward_valid(x, xdims, conv1, conv1dims, a, adims);
  dim3 DimGrid2(cdims[0], cdims[3], ceil(cdims[1]*cdims[2]/256.0));
  dim3 DimBlock2(256, 1, 1);

  const auto start3 = now();
  conv_forward_valid_kernel<<<DimGrid2, DimBlock2>>>(device_x, device_xdims, device_conv1, device_conv1dims, device_a, device_adims);
  const auto end3 = now();
  //hipMemcpy(c, device_a, sizeof(float)*device_c_size, hipMemcpyDeviceToHost);
  const auto elapsed3 = std::chrono::duration<double, std::milli>(end3 - start3).count();
  std::cout << "Done with " << bdims[0] << " queries in Conv2 "
            << "elapsed = " << elapsed3 << " milliseconds." << "\n";
  // kernel conv end

  // relu
  //relu4(c, cdims);

  // relu kernel start
  dim3 DimGrid3(ceil(device_a_size/256), 1, 1);
  dim3 DimBlock3(256, 1, 1);
  relu4_kernel<<<DimGrid3, DimBlock3>>>(device_a, device_adims);
  //hipMemcpy(c, device_a, sizeof(float)*device_c_size, hipMemcpyDeviceToHost);
  // relu kernel end

  // average pooling
  const int ddims[] = {cdims[0], cdims[1] / pool_size, cdims[2] / pool_size,
                       cdims[3]};
  int device_d_size = ddims[0]*ddims[1]*ddims[2]*ddims[3];
  auto d = zeros<float>(ddims);
  //average_pool(c, cdims, pool_size, d, ddims);

  //pooling kernel start
  hipMemcpy(device_xdims, ddims, sizeof(int)*4, hipMemcpyHostToDevice);

  hipMemcpy(device_x, d, sizeof(float)*device_d_size, hipMemcpyHostToDevice);

  w_grid = ceil(ddims[2]/tile_size);
  h_grid = ceil(ddims[1]/tile_size);

  dim3 DimGrid4(ddims[0], ddims[3], ceil(ddims[1]*ddims[2]/256.0));
  dim3 DimBlock4(256, 1, 1);
  average_pool_kernel<<<DimGrid4, DimBlock4>>>(device_a, device_adims, pool_size, device_x, device_xdims, w_grid);

  hipMemcpy(d, device_x, sizeof(float)*device_d_size, hipMemcpyDeviceToHost);
  //pooling kernel end

  // reshape
  const int ddims2[] = {ddims[0], ddims[1] * ddims[2] * ddims[3]};

  // matrix multiplication
  const int edims[] = {ddims[0], fc1dims[1]};
  auto e            = zeros<float>(edims);

  // float* deviceD;
  // int* deviceDdims2;
  float* deviceFc1;
  int* deviceFc1dims;
  // float* deviceE;
  // int* deviceEdims;

  int d2Size = ddims2[0] * ddims2[1];
  int fc1Size = fc1dims[0] * fc1dims[1];
  int eSize = edims[0] * edims[1];

  hipMalloc((void**) &deviceFc1, fc1Size * sizeof(float));
  hipMalloc((void**) &deviceFc1dims, 2 * sizeof(int));

  hipMemcpy(device_x, d, d2Size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_xdims, ddims2, 2 * sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(device_conv1, fc1, fc1Size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_conv1dims, fc1dims, 2 * sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(device_adims, edims, 2 * sizeof(int), hipMemcpyHostToDevice);

  dim3 DimGridE(ceil(edims[1]/16.0), ceil(edims[0]/16.0), 1);
  dim3 DimBlockE(16, 16, 1);
  // fully_forward(d, ddims2, fc1, fc1dims, e, edims);
  // get start time
  const auto startfc1 = now();
  fully_forward_kernel<<<DimGridE, DimBlockE>>>(device_x, device_xdims, device_conv1, device_conv1dims, device_a, device_adims);
  const auto endfc1 = now();
  const auto elapsedfc1 = std::chrono::duration<double, std::milli>(endfc1 - startfc1).count();
  std::cout << "Done with " << edims[0] << " queries in fully_forward "
           << "elapsed = " << elapsedfc1 << " milliseconds." << "\n";

  dim3 DimGridRelu2(ceil(eSize/256.0), 1, 1);
  dim3 DimBlockRelu2(256, 1, 1);
  const auto startrelu2 = now();
  relu2_kernel<<<DimGridRelu2, DimBlockRelu2>>>(device_a, device_adims);
  const auto endrelu2 = now();
  const auto elapsedrelu2 = std::chrono::duration<double, std::milli>(endrelu2 - startrelu2).count();
  std::cout << "Done with " << edims[0] << " queries in relu2 "
           << "elapsed = " << elapsedrelu2 << " milliseconds." << "\n";

  // matrix multiplication
  const int fdims[] = {edims[0], fc2dims[1]};
  auto f            = zeros<float>(fdims);

  // float* deviceFc2;
  // int* deviceFc2dims;
  // float* deviceF;
  // int* deviceFdims;

  int fc2size = fc2dims[0] * fc2dims[1];
  int fsize = fdims[0] * fdims[1];

  hipMemcpy(device_conv1, fc2, fc2size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_conv1dims, fc2dims, 2 * sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(device_xdims, fdims, 2 * sizeof(int), hipMemcpyHostToDevice);

  const auto startfc2 = now();
  //fully_forward(e, edims, fc2, fc2dims, f, fdims);
  dim3 DimGridF(ceil(fdims[1]/16.0), ceil(fdims[0]/16.0), 1);
  dim3 DimBlockF(16, 16, 1);
  fully_forward_kernel<<<DimGridF, DimBlockF>>>(device_a, device_adims, device_conv1, device_conv1dims, device_x, device_xdims);
  const auto endfc2 = now();
  const auto elapsedfc2 = std::chrono::duration<double, std::milli>(endfc2 - startfc2).count();
  std::cout << "Done with " << fdims[0] << " queries in fully_forward 2 "
           << "elapsed = " << elapsedfc2 << " milliseconds." << "\n";

  hipMemcpy(f, device_x, fsize * sizeof(float), hipMemcpyDeviceToHost);

  // // matrix multiplication
  // const int edims[] = {ddims[0], fc1dims[1]};
  // auto e            = zeros<float>(edims);
  // fully_forward(d, ddims2, fc1, fc1dims, e, edims);
  //
  // // relu
  // relu2(e, edims);
  //
  // // matrix multiplication
  // const int fdims[] = {edims[0], fc2dims[1]};
  // auto f            = zeros<float>(fdims);
  // fully_forward(e, edims, fc2, fc2dims, f, fdims);

  // device_x_size = xdims[0]*xdims[1]*xdims[2]*xdims[3];
  // device_a_size = adims[0]*adims[1]*adims[2]*adims[3];
  // device_b_size = bdims[0]*bdims[1]*bdims[2]*bdims[3];
  // device_c_size = cdims[0]*cdims[1]*cdims[2]*cdims[3];
  // device_d_size = ddims[0]*ddims[1]*ddims[2]*ddims[3];
  // int device_d2_size = ddims2[0]*ddims2[1];
  // int device_e_size = edims[0]*edims[1];
  // int device_f_size = fdims[0]*fdims[1];

  // std::cout << "more devices: " << device_x_size << " " << device_a_size << " " << device_b_size << " " << device_c_size << " " << device_d_size << " " << device_d2_size << " " << device_e_size << " " << device_f_size << std::endl;

  // argmax(f, fdims, out);

  // float* deviceF;
  // int* deviceOUT;
  // int* deviceFDIMS;
  //
  // int numInputElements = fdims[0]*fdims[1];  // number of elements in the input list
  // int numOutputElements = fdims[0];          // number of elements in the output list
  //
  // // hipMalloc((void**)&deviceF,numInputElements*sizeof(float));
  // // hipMalloc((void**)&deviceOUT,numOutputElements*sizeof(int));
  // // hipMalloc((void**)&deviceFDIMS, 2*sizeof(int));
  //
  // // hipMemcpy(deviceF, f, numInputElements*sizeof(float),hipMemcpyHostToDevice);
  // // hipMemcpy(deviceFDIMS, fdims, 2*sizeof(int), hipMemcpyHostToDevice);
  //
  // hipMemcpy(device_x, f, numInputElements*sizeof(float),hipMemcpyHostToDevice);
  // hipMemcpy(device_xdims, fdims, 2*sizeof(int), hipMemcpyHostToDevice);
  //
  // // for parallelizing inner loop
  // // dim3 dimGrid(10, 1, 1);
  // // dim3 dimBlock(32, 1, 1);
  //
  // // for paralleliziing outer loop
  // dim3 dimGrid(ceil(fdims[0]/1024.0), 1, 1);
  // dim3 dimBlock(1024, 1, 1);
  //
  //

  dim3 DimGrid9((FLAGS_batch_size - 1)/16 + 1, 1, 1);
  dim3 DimBlock9(16, 1, 1);

  const auto start_arg = now();
  argmax_kernel<<<DimGrid9,DimBlock9>>>(device_x,device_xdims,(int*)device_a);
  // argmax_kernel<<<DimGrid9,DimBlock9>>>(deviceF,deviceFDIMS,deviceOUT);
  const auto end_arg = now();
  const auto elapsed_arg = std::chrono::duration<double, std::milli>(end_arg - start_arg).count();
  std::cout << "Done with argmax "
          << "elapsed = " << elapsed << " milliseconds." << "\n";
  
  hipMemcpy(out, device_a, FLAGS_batch_size * sizeof(int), hipMemcpyDeviceToHost);

  // hipFree(deviceF);
  // hipFree(deviceOUT);
  // hipFree(deviceFDIMS);

  hipFree(device_x);
  hipFree(device_xdims);
  hipFree(device_a);
  hipFree(device_adims);
  hipFree(device_conv1);
  hipFree(device_conv1dims);
  hipFree(deviceFc1);
  hipFree(deviceFc1dims);

  delete[] a;
  delete[] b;
  delete[] c;
  delete[] d;
  delete[] e;
  delete[] f;
}

int main(int argc, char **argv) {

  if (argc != 3 && argc != 4) {
    std::cerr << "\n"
              << "This program performs the forward opertion step for "
                 "Convolutional Neural Network(CNN).  "
                 "Sample usage: \n"
              << argv[0]
              << " [../data/test10.hdf5] [../data/model.hdf5] [10]\n";
    return -1;
  }
  FLAGS_testdata = std::string(argv[1]);
  FLAGS_model    = std::string(argv[2]);
  if (argc == 3) {
    const std::map<std::string, int> default_batch_sizes{
        {"../data/test2.hdf5", 2},
        {"../data/test10.hdf5", 10},
        {"../data/test100.hdf5", 100},
        {"../data/testfull.hdf5", 10000}};
    const auto batch_size_in_map = default_batch_sizes.find(FLAGS_testdata);
    if (batch_size_in_map == default_batch_sizes.end()) {
      std::cerr << "\nERROR:: Unrecognized file " << FLAGS_testdata << " batch_size must be specified.\n";
      return -1;
    }
    FLAGS_batch_size = batch_size_in_map->second;
  } else if (argc == 4) {
    FLAGS_batch_size = atoi(argv[3]);
  }
  xdims[0] = FLAGS_batch_size;
  rdims[0] = FLAGS_batch_size;

  // Load data into x and y
  float *x = allocate<float>(xdims);
  float *y = allocate<float>(rdims);
  loadData(x, y);

  // Load model
  float *conv1 = allocate<float>(conv1dims);
  float *conv2 = allocate<float>(conv2dims);
  float *fc1   = allocate<float>(fc1dims);
  float *fc2   = allocate<float>(fc2dims);
  loadModel(conv1, conv2, fc1, fc2);

  // Perform foward opertion
  int *out = zeros<int>(FLAGS_batch_size);

  // get start time
  const auto start = now();

  forward_operation(x, conv1, conv2, fc1, fc2, out);

  // get end time
  const auto end = now();

  // get elapsed time in milliseconds
  const auto elapsed =
      std::chrono::duration<double, std::milli>(end - start).count();

  // Get reference
  int *ref = zeros<int>(FLAGS_batch_size);
  argmax(y, rdims, ref);

  // Calculate correctness
  int num_correct = 0;
  for (const auto i : range(0, FLAGS_batch_size)) {
    if (out[i] == ref[i]) {
      num_correct++;
    }
  }
  std::cout << "Done with " << FLAGS_batch_size << " queries in "
            << "elapsed = " << elapsed << " milliseconds. Correctness: "
            << static_cast<float>(num_correct) / FLAGS_batch_size << "\n";

  delete[] x;
  delete[] y;
  delete[] conv1;
  delete[] conv2;
  delete[] fc1;
  delete[] fc2;
  delete[] out;
  delete[] ref;

  return 0;
}
